#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cmath>
#include "procImg.h" // Donde se declara Pixel y la funcion procImg(...)



////////////////////////////////////////////////////////////////////////////////
// Kernel: Convierte un pixel a escala de grises
////////////////////////////////////////////////////////////////////////////////
__global__ void toGrayKernel(Pixel* d_pixels, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int idx = y * width + x;
        float r = static_cast<float>(d_pixels[idx].r);
        float g = static_cast<float>(d_pixels[idx].g);
        float b = static_cast<float>(d_pixels[idx].b);

        float grayf = 0.299f * r + 0.587f * g + 0.114f * b;
        unsigned char gray = static_cast<unsigned char>(grayf);

        d_pixels[idx].r = gray;
        d_pixels[idx].g = gray;
        d_pixels[idx].b = gray;
    }
}

////////////////////////////////////////////////////////////////////////////////
// Kernel: Invierte colores
// R = 255 - R, G = 255 - G, B = 255 - B
////////////////////////////////////////////////////////////////////////////////
__global__ void invertColorsKernel(Pixel* d_pixels, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        int idx = y * width + x;
        d_pixels[idx].r = 255 - d_pixels[idx].r;
        d_pixels[idx].g = 255 - d_pixels[idx].g;
        d_pixels[idx].b = 255 - d_pixels[idx].b;
    }
}

// Kernel: Pixelado con filtro cuadrado de tamFiltro x tamFiltro
// Cada hilo copia no solo su pixel central, sino tambien parte del halo.
// Sin padding artificial: si la ventana se sale de [0..width-1], [0..height-1], no aporta nada (no se suma).
__global__ void pixelateKernel(const Pixel* d_in, Pixel* d_out,
    int width, int height, int tamFiltro)
{
    // Coordenadas globales del píxel
    int gx = blockIdx.x * blockDim.x + threadIdx.x;
    int gy = blockIdx.y * blockDim.y + threadIdx.y;
    bool inside = (gx < width) && (gy < height);

    // Memoria compartida para la tesela y la reducción
    extern __shared__ int shared_mem[];
    Pixel* sData = (Pixel*)shared_mem;
    int* sR = (int*)(sData + blockDim.x * blockDim.y);
    int* sG = sR + blockDim.x * blockDim.y;
    int* sB = sG + blockDim.x * blockDim.y;
    int* sCount = sB + blockDim.x * blockDim.y;

    int tid = threadIdx.y * blockDim.x + threadIdx.x;

    // Cargar píxel en memoria compartida
    if (inside) {
        sData[tid] = d_in[gy * width + gx];
    }
    else {
        sData[tid] = { 0, 0, 0 };
    }

    // Inicializar acumuladores
    sR[tid] = (inside) ? sData[tid].r : 0;
    sG[tid] = (inside) ? sData[tid].g : 0;
    sB[tid] = (inside) ? sData[tid].b : 0;
    sCount[tid] = (inside) ? 1 : 0;

    __syncthreads();

    // Reducción paralela para sumar R, G, B y contar píxeles válidos
    for (int stride = blockDim.x * blockDim.y / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            sR[tid] += sR[tid + stride];
            sG[tid] += sG[tid + stride];
            sB[tid] += sB[tid + stride];
            sCount[tid] += sCount[tid + stride];
        }
        __syncthreads();
    }

    // Thread 0 calcula el promedio y lo almacena en sData[0]
    if (tid == 0) {
        Pixel avg;
        if (sCount[0] > 0) {
            avg.r = (unsigned char)(sR[0] / sCount[0]);
            avg.g = (unsigned char)(sG[0] / sCount[0]);
            avg.b = (unsigned char)(sB[0] / sCount[0]);
        }
        else {
            avg = { 0, 0, 0 };
        }
        sData[0] = avg;
    }

    __syncthreads();

    // Todos los hilos escriben el promedio en su píxel
    if (inside) {
        d_out[gy * width + gx] = sData[0];
    }
}
// para la carga completa del halo.
int procImg(Pixel* pixels, int height, int width, int option, int tamFiltro)
{
    hipError_t cudaStatus;
    Pixel* d_in = nullptr;
    Pixel* d_out = nullptr;
    int totalPixels = width * height;

    // 1) Info de la GPU
    int devID = 0;
    hipDeviceProp_t prop;
    cudaStatus = hipGetDeviceProperties(&prop, devID);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGetDeviceProperties failed!\n");
        return 1;
    }

    // 2) Determinar blockDim.x = blockDim.y = bCandidate
    int maxThreads = prop.maxThreadsPerBlock;
    int bCandidate = static_cast<int>(floorf(std::sqrtf((float)maxThreads)));
    if (bCandidate > prop.maxThreadsDim[0]) bCandidate = prop.maxThreadsDim[0];
    if (bCandidate > prop.maxThreadsDim[1]) bCandidate = prop.maxThreadsDim[1];
    dim3 blockDim(bCandidate, bCandidate, 1);

    // 3) Calcular grid
    int gridX = (width + blockDim.x - 1) / blockDim.x;
    int gridY = (height + blockDim.y - 1) / blockDim.y;
    dim3 gridDim(gridX, gridY, 1);

    printf("=== GPU Info ===\n");
    printf("Device: %s\n", prop.name);
    printf("Compute Capability: %d.%d\n", prop.major, prop.minor);
    printf("MaxThreadsPerBlock: %d, blockDim=(%d,%d), gridDim=(%d,%d)\n",
        maxThreads, blockDim.x, blockDim.y, gridDim.x, gridDim.y);
    printf("tamFiltro = %d\n", tamFiltro);
    printf("================\n");

    // 4) Seleccionar device
    cudaStatus = hipSetDevice(devID);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!\n");
        return 1;
    }

    // 5) Reservar memoria en GPU
    cudaStatus = hipMalloc((void**)&d_in, totalPixels * sizeof(Pixel));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc d_in failed!\n");
        return 1;
    }
    cudaStatus = hipMalloc((void**)&d_out, totalPixels * sizeof(Pixel));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc d_out failed!\n");
        hipFree(d_in);
        return 1;
    }

    // 6) Copiar la imagen host->device
    cudaStatus = hipMemcpy(d_in, pixels, totalPixels * sizeof(Pixel), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy H->D failed!\n");
        hipFree(d_in);
        hipFree(d_out);
        return 1;
    }

    // 7) Segun la opcion
    switch (option) {
    case 1: // Blanco y Negro (in-place)
        toGrayKernel << <gridDim, blockDim >> > (d_in, width, height);
        break;

    case 6: // Invertir colores (in-place)
        invertColorsKernel << <gridDim, blockDim >> > (d_in, width, height);
        break;

    case 31: // Pixelar color
    {
        /*int radius = tamFiltro / 2;
        int tileW = blockDim.x + 2 * radius;
        int tileH = blockDim.y + 2 * radius;*/
        /*size_t needed = tileW * tileH * sizeof(Pixel);
        size_t needed = blockSize.x * blockSize.y * (sizeof(Pixel) + 4 * sizeof(int));*/
        // Calcular la memoria compartida necesaria para pixelateKernel
        int nThreads = blockDim.x * blockDim.y;

        // Tamaño para los pixeles
        size_t sizeData = nThreads * sizeof(Pixel);

        // Tamaño para R, G, B, Count (4 int arrays)
        size_t sizeR = nThreads * sizeof(int);
        size_t sizeG = nThreads * sizeof(int);
        size_t sizeB = nThreads * sizeof(int);
        size_t sizeCount = nThreads * sizeof(int);

        // Total
        size_t totalNeeded = sizeData + sizeR + sizeG + sizeB + sizeCount;


        if (totalNeeded > prop.sharedMemPerBlock) {
            fprintf(stderr, "Error: tamFiltro=%d => %zu bytes en shared, pero solo hay %zu.\n",
                tamFiltro, totalNeeded, (size_t)prop.sharedMemPerBlock);
            hipFree(d_in); hipFree(d_out);
            return 1;
        }

        pixelateKernel << <gridDim, blockDim, totalNeeded >> > (d_in, d_out, width, height, tamFiltro);
    }
    break;

    case 32: // Pixelar BN (1) BN in-place, (2) pixelar out-of-place
    {
        toGrayKernel << <gridDim, blockDim >> > (d_in, width, height);
        hipDeviceSynchronize();

        int nThreads = blockDim.x * blockDim.y;

        // Tamaño para los pixeles
        size_t sizeData = nThreads * sizeof(Pixel);

        // Tamaño para R, G, B, Count (4 int arrays)
        size_t sizeR = nThreads * sizeof(int);
        size_t sizeG = nThreads * sizeof(int);
        size_t sizeB = nThreads * sizeof(int);
        size_t sizeCount = nThreads * sizeof(int);

        // Total
        size_t totalNeeded = sizeData + sizeR + sizeG + sizeB + sizeCount;

        if (totalNeeded > prop.sharedMemPerBlock) {
            fprintf(stderr, "Error: tamFiltro=%d => %zu bytes en shared, solo hay %zu.\n",
                tamFiltro, totalNeeded, (size_t)prop.sharedMemPerBlock);
            hipFree(d_in); hipFree(d_out);
            return 1;
        }

        pixelateKernel << <gridDim, blockDim, totalNeeded >> > (d_in, d_out, width, height, tamFiltro);
    }
    break;

    default:
        fprintf(stderr, "Opcion %d no reconocida.\n", option);
        hipFree(d_in);
        hipFree(d_out);
        return 1;
    }

    // Verificar si hubo error lanzando el kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_in); hipFree(d_out);
        return 1;
    }

    // Sincronizar
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize error %d\n", cudaStatus);
        hipFree(d_in); hipFree(d_out);
        return 1;
    }

    // 8) Copiar el resultado de vuelta
    bool usedPixelate = (option == 31 || option == 32);
    Pixel* d_result = usedPixelate ? d_out : d_in;
    cudaStatus = hipMemcpy(pixels, d_result, totalPixels * sizeof(Pixel), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy D->H failed!\n");
        hipFree(d_in);
        hipFree(d_out);
        return 1;
    }

    // 9) Liberar
    hipFree(d_in);
    hipFree(d_out);

    // 10) (Opcional) reset
    hipDeviceReset();

    return 0; // exito
}
