#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <iostream>
#include <vector>
#include "procImg.h"

#include <fstream>
#include <cstdint>
#include <cstdlib>
#include <cstdio>

#pragma pack(push, 1)
struct BMPHeader_t {
    uint16_t  type;
    uint32_t  size;
    uint16_t  reserved1;
    uint16_t  reserved2;
    uint32_t  offset;
    uint32_t  dib_header_size;
    int32_t   width_px;
    int32_t   height_px;
    uint16_t  num_planes;
    uint16_t  bits_per_pixel;
    uint32_t  compression;
    uint32_t  image_size_bytes;
    int32_t   x_resolution_ppm;
    int32_t   y_resolution_ppm;
    uint32_t  num_colors;
    uint32_t  important_colors;
};
#pragma pack(pop)

struct BMPImage_t {
    BMPHeader_t header;
    uint8_t* data;
};

#define MAGIC_VALUE         0x4D42
#define BITS_PER_PIXEL      24
#define BYTES_PER_PIXEL     (BITS_PER_PIXEL / 8)

static int getPadding(const BMPHeader_t* header) {
    return (4 - (header->width_px * BYTES_PER_PIXEL) % 4) % 4;
}
static int getRowSize(const BMPHeader_t* header) {
    return header->width_px * BYTES_PER_PIXEL + getPadding(header);
}
static int getPosition(const BMPHeader_t* header, int x, int y) {
    int stride = getRowSize(header);
    int j = (header->height_px - y - 1) * stride;
    int i = x * BYTES_PER_PIXEL;
    return j + i;
}
static int CheckHeader(const BMPHeader_t* header) {
    return header->type == MAGIC_VALUE &&
        header->bits_per_pixel == BITS_PER_PIXEL;
}

// Leer BMP
BMPImage_t* ReadBMP(const char* filename) {
    BMPImage_t* bmp = (BMPImage_t*)malloc(sizeof(BMPImage_t));
    FILE* fp = fopen(filename, "rb");
    if (!fp) {
        printf("Error abriendo archivo BMP: %s\n", filename);
        exit(EXIT_FAILURE);
    }
    fread(&bmp->header, sizeof(BMPHeader_t), 1, fp);
    if (!CheckHeader(&bmp->header)) {
        printf("Cabecera BMP no valida.\n");
        exit(EXIT_FAILURE);
    }
    bmp->data = (uint8_t*)malloc(bmp->header.image_size_bytes);
    fseek(fp, bmp->header.offset, SEEK_SET);
    fread(bmp->data, bmp->header.image_size_bytes, 1, fp);
    fclose(fp);
    return bmp;
}

// Guardar BMP
void SaveBMP(BMPImage_t* bmp, const char* filename) {
    FILE* fp = fopen(filename, "wb");
    if (!fp) {
        printf("Error guardando archivo BMP: %s\n", filename);
        return;
    }
    fwrite(&bmp->header, sizeof(BMPHeader_t), 1, fp);
    fseek(fp, bmp->header.offset, SEEK_SET);
    fwrite(bmp->data, bmp->header.image_size_bytes, 1, fp);
    fclose(fp);
}

// Liberar
void DestroyBMP(BMPImage_t* bmp) {
    if (bmp) {
        free(bmp->data);
        free(bmp);
    }
}

// Convierte bmp->data (con padding) a un array lineal de Pixel (sin padding)
std::vector<Pixel> bmpToPixelArray(const BMPImage_t* bmp) {
    int width = bmp->header.width_px;
    int height = bmp->header.height_px;
    std::vector<Pixel> out;
    out.reserve(width * height);
    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            int pos = getPosition(&bmp->header, x, y);
            Pixel p;
            p.b = bmp->data[pos + 0];
            p.g = bmp->data[pos + 1];
            p.r = bmp->data[pos + 2];
            out.push_back(p);
        }
    }
    return out;
}

// Copia de vuelta
void pixelArrayToBMP(BMPImage_t* bmp, const std::vector<Pixel>& in) {
    int width = bmp->header.width_px;
    int height = bmp->header.height_px;
    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            int pos = getPosition(&bmp->header, x, y);
            int idx = y * width + x;
            bmp->data[pos + 0] = in[idx].b;
            bmp->data[pos + 1] = in[idx].g;
            bmp->data[pos + 2] = in[idx].r;
        }
    }
}

int main() {
    char ruta[256];
    memset(ruta, 0, sizeof(ruta));
    const char* defaultFile = "test.bmp";
    printf("Introduzca la ruta de la imagen BMP (Intro para usar '%s'): ", defaultFile);
    if (fgets(ruta, sizeof(ruta), stdin) == nullptr) {
        strcpy(ruta, defaultFile);
    }
    else {
        size_t len = strlen(ruta);
        if (len > 0 && ruta[len - 1] == '\n')
            ruta[len - 1] = '\0';
        if (strlen(ruta) == 0)
            strcpy(ruta, defaultFile);
    }

    printf("Leyendo archivo: '%s'\n", ruta);
    BMPImage_t* bmp = ReadBMP(ruta);
    int width = bmp->header.width_px;
    int height = bmp->header.height_px;
    printf("Imagen de %d x %d pixeles\n", width, height);

    std::vector<Pixel> pixels = bmpToPixelArray(bmp);

    printf("Opciones\n"
        "  (1) Conversion a Blanco y Negro\n"
        "  (2) Pixelar\n"
        "  (3) Identificar colores (sin halo)\n"
        "  (4) Filtro y delineado de zonas de color\n"
        "  (5) Calculo de pseudo-hash\n"
        "  (6) Invertir colores\n"
        "  (X) Salir\n");
    printf("Elija opcion: ");

    int option;
    scanf("%d", &option);

    int filterDiv = 1;
    const char* outName = nullptr;

    switch (option) {
    case 1:
        procImg(pixels.data(), height, width, 1, filterDiv, nullptr, 0);
        outName = "out_gray.bmp";
        break;

    case 2: {
        printf("Has elegido pixelar: \n (1) Color \n (2) Blanco y Negro\n");
        int subSel;
        scanf("%d", &subSel);
        printf("Introduce factor de division del blockDim: ");
        scanf("%d", &filterDiv);
        if (subSel == 1) {
            procImg(pixels.data(), height, width, 21, filterDiv, nullptr, 0);
            outName = "out_pixel_color.bmp";
        }
        else if (subSel == 2) {
            procImg(pixels.data(), height, width, 22, filterDiv, nullptr, 0);
            outName = "out_pixel_bn.bmp";
        }
        else {
            printf("Seleccion no valida. Saliendo...\n");
            DestroyBMP(bmp);
            return 0;
        }
    }
          break;

    case 3: {
        // Identificaci�n sin halo
        unsigned int count = 0;
        printf("Elija el color a identificar (1: Rojo, 2: Verde, 3: Azul): ");
        int colorSel;
        scanf("%d", &colorSel);
        int optIdent;
        if (colorSel == 1)
            optIdent = 31;
        else if (colorSel == 2)
            optIdent = 32;
        else if (colorSel == 3)
            optIdent = 33;
        else {
            printf("Color no valido. Saliendo...\n");
            DestroyBMP(bmp);
            return 0;
        }
        procImg(pixels.data(), height, width, optIdent, 0, &count, 0);
        if (colorSel == 1)
            outName = "out_red.bmp";
        else if (colorSel == 2)
            outName = "out_green.bmp";
        else
            outName = "out_blue.bmp";
        printf("Num pixeles identificados = %u\n", count);
    }
          break;

    case 4: {
        // Filtro y delineado de zonas de color: se usan las opciones 41,42,43 según el color
        printf("Has elegido filtro y delineado.\n");
        printf("Elija el color a delinear (1: Rojo, 2: Verde, 3: Azul): ");
        int colorChoice;
        scanf("%d", &colorChoice);
        int optDelineado;
        if (colorChoice == 1)
            optDelineado = 41;
        else if (colorChoice == 2)
            optDelineado = 42;
        else if (colorChoice == 3)
            optDelineado = 43;
        else {
            printf("Color no valido. Saliendo...\n");
            DestroyBMP(bmp);
            return 0;
        }
        printf("Introduce factor de division del blockDim: ");
        scanf("%d", &filterDiv);
        printf("Introduzca el tama�o del halo: ");
        int haloSize;
        scanf("%d", &haloSize);
        procImg(pixels.data(), height, width, optDelineado, filterDiv, nullptr, haloSize);
        outName = "out_filtro_delineado.bmp";
    }
          break;

    case 5: {
        // Pseudo-hash: calcular y mostrar el hash en consola
        // En esta opci�n, procImg realiza la reducci�n y muestra el hash textual.
        procImg(pixels.data(), height, width, 5, filterDiv, nullptr, 0);
        // No se guarda imagen; se asume que el hash se muestra por consola.
        printf("Hash calculado. Consulte la salida en consola.\n");
        // Salir inmediatamente
        DestroyBMP(bmp);
        return 0;
    }
          break;

    case 6: {
        procImg(pixels.data(), height, width, 6, filterDiv, nullptr, 0);
        outName = "out_inverted.bmp";
    }
        break;

    default:
        printf("Opcion no valida. Saliendo...\n");
        DestroyBMP(bmp);
        return 0;
    }

    if (!outName) {
        outName = "out_result.bmp";
    }

    pixelArrayToBMP(bmp, pixels);
    SaveBMP(bmp, outName);
    printf("Guardado resultado en: %s\n", outName);

    DestroyBMP(bmp);
    return 0;
}
